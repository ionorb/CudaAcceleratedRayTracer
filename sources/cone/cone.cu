#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/04/07 18:24:41 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/07 18:24:54 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_discr	get_cone_disc(t_vec *f_n, double tan)
{
	t_discr	ret;

	ret.a = int_pow(f_n[1].x, 2) + int_pow(f_n[1].y, 2) \
	- int_pow((f_n[1].z * tan), 2);
	ret.b = 2 * ((f_n[0].x * f_n[1].x) + (f_n[0].y * f_n[1].y) \
	- (f_n[0].z * f_n[1].z * int_pow(tan, 2)));
	ret.c = int_pow(f_n[0].x, 2) + int_pow(f_n[0].y, 2) \
	- int_pow((f_n[0].z * tan), 2);
	ret.dscr = int_pow(ret.b, 2) - (4 * (ret.a * ret.c));
	return (ret);
}

int	is_in_top(double *curr_tan, t_cone cone, t_vec new_cam, t_vec new_dir)
{
	double	radius;

	radius = cone.height * curr_tan[1];
	if (int_pow(new_cam.x + (curr_tan[0] * new_dir.x), 2)
		+ int_pow(new_cam.y + (curr_tan[0] * new_dir.y), 2)
		<= int_pow(radius, 2))
		return (1);
	return (0);
}

t_cuad_ctr	check_cone_body(t_cuad_ctr *crr, t_cone cone, \
t_vec *f_n, double tan)
{
	t_cuad_ctr	ret;
	t_discr		discr;
	double		curr;
	t_vec		h_ctrl;

	ret = *crr;
	discr = get_cone_disc(f_n, tan);
	if (discr.dscr >= 0.0)
	{
		curr = solve_cone_quad(&discr, f_n);
		if (curr > 0 && (ret.c == -1 || curr < ret.c))
		{
			h_ctrl = vec_sum(f_n[0], scal_vec(curr, f_n[1]));
			if (0 <= h_ctrl.z && h_ctrl.z <= cone.height)
			{
				ret.c = curr;
				ret.cap_ctrl = 2;
			}
		}
	}
	return (ret);
}

t_cuad_ctr	get_dist_to_cone(t_cone cone, t_vec n_c, t_vec n_d, double tang)
{
	t_cuad_ctr	ret;
	double		curr_tan[2];
	t_vec		fuck_normntte[2];

	curr_tan[1] = tang;
	ret.c = -1;
	ret.cap_ctrl = 0;
	if (v_abs(n_d.z) > 0.0001)
	{
		curr_tan[0] = ((cone.height / n_d.z) - (n_c.z / n_d.z));
		if (curr_tan[0] > 0 && is_in_top(curr_tan, cone, n_c, n_d)
			&& (ret.c == -1 || curr_tan[0] < ret.c))
		{
			ret.c = curr_tan[0];
			ret.cap_ctrl = 1;
		}
	}
	fuck_normntte[0] = n_c;
	fuck_normntte[1] = n_d;
	ret = check_cone_body(&ret, cone, fuck_normntte, curr_tan[1]);
	return (ret);
}

void	check_cones(t_mrt *mrt, t_inter *ctrl, t_vec point, t_vec dir)
{
	int				i;
	t_cuad_ctr		ctr;
	t_vec			newy[2];
	double			tang;
	t_mtrx			chg_base;

	i = -1;
	while (++i < mrt->obj_count[CONE])
	{
		tang = tan(mrt->cone[i].angle / 2);
		newy[0] = vec_rest(point, mrt->cone[i].pos);
		chg_base = fill_mtrx(mrt->cone[i].base.n1, mrt->cone[i].base.n2,
				mrt->cone[i].base.n3);
		newy[0] = mtrx_by_vec(chg_base, newy[0]);
		newy[1] = mtrx_by_vec(chg_base, dir);
		ctr = get_dist_to_cone(mrt->cone[i], newy[0], newy[1], tang);
		if (ctr.c > 0 && (ctrl->dist == -1 || ctr.c < ctrl->dist))
		{
			*ctrl = (t_inter){CONE, i, ctr.c, vec_sum(point, \
			scal_vec(ctr.c, dir)), fill_coord(0, 0, 0), \
			get_cone_color(mrt, i, newy, ctr), mrt->cone[i].option, \
			ctr.cap_ctrl, cam_in_cone(mrt, i, newy[0], tang)};
			// ctrl->color = get_cone_texture(mrt, *ctrl);
		}
	}
}
