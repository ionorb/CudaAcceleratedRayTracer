#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone_color.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/04/05 07:27:52 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/07 18:42:19 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_c_chess	get_cone_bdy_col(t_mrt *mrt, int i, t_vec coor, t_rgb color)
{
	double		aux_dist;
	t_c_chess	ret;

	ret.color = color;
	ret.even_ctrl = 0;
	aux_dist = integer_part((mrt->cone[i].height
				* int_pow(2, mrt->cone[i].option.chess_ctrl))
			/ (PI * mrt->cone[i].height * tan(mrt->cone[i].angle / 2)));
	if ((int)aux_dist % 2 == 1)
		aux_dist += 1.0;
	aux_dist = mrt->cone[i].height / aux_dist;
	if (((int)v_abs(integer_part(int_pow(2, mrt->cone[i].option.chess_ctrl) \
		* (coor.y / PI))) % 2 == 1 && (int)integer_part(v_abs(coor.z \
		- mrt->cone[i].height) / aux_dist) % 2 == 1) \
		|| (((int)v_abs(integer_part(int_pow(2, \
		mrt->cone[i].option.chess_ctrl) * (coor.y / PI))) % 2 \
		== 0 && (int)integer_part(v_abs(coor.z - mrt->cone[i].height) / \
		aux_dist) % 2 == 0)))
	{
		ret.color = mrt->cone[i].option.check_color;
		ret.even_ctrl = (int)integer_part(mrt->cone[i].height \
		/ aux_dist) % 2;
	}
	return (ret);
}

t_rgb	get_cone_color(t_mrt *mrt, int i, t_vec *newy, t_cuad_ctr ctr)
{
	t_vec		new_inter;
	t_rgb		ret;
	t_c_chess	ctrl;

	ret = mrt->cone[i].color;
	if (mrt->cone[i].option.chess_ctrl > 0)
	{
		new_inter = get_cyl_coor(vec_sum(newy[0], scal_vec(ctr.c, newy[1])));
		ctrl = get_cone_bdy_col(mrt, i, new_inter, ret);
		if (ctr.cap_ctrl == 2)
			ret = ctrl.color;
		if ((ctr.cap_ctrl == 1)
			&& ((int)v_abs(integer_part(
						int_pow(2, mrt->cone[i].option.chess_ctrl)
						* (new_inter.y / PI)))) % 2
			== (ctrl.even_ctrl + ctr.cap_ctrl) % 2)
			ret = mrt->cone[i].option.check_color;
	}
	return (ret);
}
