#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone_utils3.c                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: yridgway <yridgway@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/04/05 07:27:52 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/07 17:03:36 by yridgway         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_vec	cone_cap_nrml_fr_map(t_mrt *mrt, t_inter inter, t_vec c_cr, \
t_c_bump_val values)
{
	int		bump_coor[2];
	t_vec	ret;

	ret = fill_coord(0, 0, 1);
	if (values.i_diam > 6)
	{
		bump_coor[0] = (mrt->cone[inter.index].option.bump_map.height / 2) \
		+ (int)integer_part(c_cr.x / values.res_cap);
		bump_coor[1] = (mrt->cone[inter.index].option.bump_map.width / 2) \
		+ ((int)integer_part(c_cr.y / values.res_cap));
		pthread_mutex_lock(mrt->mutexs);
		ret = bump_nrml_by_coor(&mrt->cone[inter.index].option, \
		bump_coor[0], bump_coor[1], 0.4);
		pthread_mutex_unlock(mrt->mutexs);
	}
	return (ret);
}

static double	get_cone_ang_resol(t_mrt *mrt, t_inter inter, \
double r_c, t_vec cyl_c)
{
	double	arc;
	int		i_res_c;
	int		n_img;
	double	ret;

	ret = r_c;
	arc = r_c / (cyl_c.y * tan(mrt->cone[inter.index].angle / 2));
	i_res_c = (int)((2 * PI) / arc);
	n_img = i_res_c / (mrt->cone[inter.index].option.bump_map.width - 1);
	if (n_img <= 0)
		ret = (2 * PI) / (mrt->cone[inter.index].option.bump_map.width - 1);
	else
	{
		if (decimal_part((2 * PI) / n_img) <= 0.5)
			ret = (2 * PI) / (n_img * i_res_c);
		else
			ret = (2 * PI) / ((n_img + 1) * i_res_c);
	}
	return (ret);
}

static double	get_cone_bdy_res(t_mrt *mrt, t_inter inter, double r_c)
{
	int		i_res_h;
	int		n_img;
	double	ret;

	ret = r_c;
	i_res_h = (int)integer_part(mrt->cone[inter.index].height / r_c);
	n_img = i_res_h / (mrt->cone[inter.index].option.bump_map.height - 1);
	if (n_img <= 0)
		ret = mrt->cone[inter.index].height / \
		(mrt->cone[inter.index].option.bump_map.height - 1);
	else
	{
		if (decimal_part(mrt->cone[inter.index].height / n_img) <= 0.5)
			ret = mrt->cone[inter.index].height / (n_img * i_res_h);
		else
			ret = mrt->cone[inter.index].height / ((n_img + 1) * i_res_h);
	}
	return (ret);
}

t_vec	cone_body_nrml_fr_map(t_mrt *mrt, t_inter inter, t_vec cyl_c, \
t_c_bump_val values)
{
	int		bump_coor[2];
	double	res_circ;
	double	res_h;
	t_vec	ret;

	ret = fill_coord(0, 0, 1);
	res_circ = get_cone_ang_resol(mrt, inter, values.res_cap, cyl_c);
	res_h = get_cone_bdy_res(mrt, inter, values.res_cap);
	bump_coor[1] = (int)integer_part(cyl_c.y / res_circ) % \
	mrt->cone[inter.index].option.bump_map.width - 1;
	bump_coor[0] = (mrt->cone[inter.index].option.bump_map.height - 1) \
	- (int)integer_part((cyl_c.z + ((mrt->cone[inter.index].height) / 2)) \
	/ res_h) % (mrt->cone[inter.index].option.bump_map.height - 1);
	pthread_mutex_lock(mrt->mutexs);
	ret = bump_nrml_by_coor(&mrt->cone[inter.index].option, \
	bump_coor[0], bump_coor[1], 0.4);
	pthread_mutex_unlock(mrt->mutexs);
	return (ret);
}

t_vec	cone_nml_frm_map(t_mrt *mrt, t_inter inter, t_vec c_cr, t_vec cyl_cr)
{
	t_vec			ret;
	t_c_bump_val	values;

	values.i_diam = i_min_v(mrt->cone[inter.index].option.bump_map.height,
			mrt->cone[inter.index].option.bump_map.width);
	values.res_cap = (2 * mrt->cone[inter.index].height * \
	tan(mrt->cone[inter.index].angle / 2)) / (values.i_diam);
	ret = fill_coord(0, 0, 1);
	if (inter.cuad_ctr == 1)
		ret = cone_cap_nrml_fr_map(mrt, inter, c_cr, values);
	if (inter.cuad_ctr == 2)
		ret = cone_body_nrml_fr_map(mrt, inter, cyl_cr, values);
	return (ret);
}
