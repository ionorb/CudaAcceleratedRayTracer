#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone_utils.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/04/07 18:25:09 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/07 18:25:24 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_vec	normal_cyl_body(t_mrt *mrt, t_inter inter)
{
	t_vec	ret;
	t_vec	newy;
	t_vec	curr;
	t_mtrx	chg_base;

	chg_base = fill_mtrx(mrt->cone[inter.index].base.n1,
			mrt->cone[inter.index].base.n2,
			mrt->cone[inter.index].base.n3);
	newy = vec_rest(inter.inter_coor, mrt->cone[inter.index].pos);
	newy = mtrx_by_vec(chg_base, newy);
	curr = fill_coord(0, 0, newy.z * \
	(1 + int_pow(tan(mrt->cone[inter.index].angle / 2), 2)));
	ret = normalize(vec_rest(newy, curr));
	ret = mtrx_by_vec(mtrx_trsp(chg_base), ret);
	return (ret);
}

t_vec	get_normal_cone(t_mrt *mrt, t_inter inter)
{
	t_vec	ret;

	ret = fill_coord(0, 0, 0);
	if (inter.cuad_ctr == 1)
		ret = mrt->cone[inter.index].dir;
	else if (inter.cuad_ctr == 2)
		ret = normal_cyl_body(mrt, inter);
	if (inter.is_in_obj)
		ret = scal_vec(-1, ret);
	if (mrt->cone[inter.index].option.b_mp_ctrl == 1)
		ret = cone_bumped(mrt, inter, ret);
	return (ret);
}

double	solve_cone_quad(t_discr *info, t_vec *f_n)
{
	double	op1;
	double	op2;
	double	ret;

	if (info->dscr < 0.00001)
		return (-info->b / (2 * info->a));
	op1 = (-info->b + sqrt(info->dscr)) / (2 * info->a);
	op2 = (-info->b - sqrt(info->dscr)) / (2 * info->a);
	ret = min_v(op1, op2);
	if (ret < 0 || vec_sum(f_n[0], scal_vec(ret, f_n[1])).z < 0)
		return (max_v(op1, op2));
	return (ret);
}

int	cam_in_cone(t_mrt *mrt, int indx, t_vec n_c, double tan)
{
	double	radius;

	if (n_c.z >= 0 && n_c.z < mrt->cone[indx].height)
	{
		radius = n_c.z * tan;
		if (int_pow(n_c.x, 2) + int_pow(n_c.y, 2)
			< int_pow(radius, 2))
			return (1);
	}
	return (0);
}

t_rgb	check_cone_contour(t_mrt *mrt, t_vec curr_dir, t_rgb color)
{
	t_cuad_ctr		ctr;
	t_vec			newy[2];
	t_mtrx			chg_base;
	double			tang;

	tang = tan(mrt->cone[mrt->curr_obj.index].angle / 2);
	newy[0] = vec_rest(mrt->cam.pos, mrt->cone[mrt->curr_obj.index].pos);
	chg_base = fill_mtrx(mrt->cone[mrt->curr_obj.index].base.n1,
			mrt->cone[mrt->curr_obj.index].base.n2,
			mrt->cone[mrt->curr_obj.index].base.n3);
	newy[0] = mtrx_by_vec(chg_base, newy[0]);
	newy[1] = mtrx_by_vec(chg_base, curr_dir);
	ctr = get_dist_to_cone(mrt->cone[mrt->curr_obj.index], newy[0], \
	newy[1], tang);
	if (ctr.c > 0)
	{
		if ((ctr.cap_ctrl == 1 && v_abs(int_pow(newy[0].x + \
		(ctr.c * newy[1].x), 2) + int_pow(newy[0].y + (ctr.c * \
		newy[1].y), 2) - int_pow(mrt->cone[mrt->curr_obj.index].height * \
		tang, 2)) < 0.009 * ctr.c * mrt->cone[mrt->curr_obj.index].height * \
		tang) || (ctr.cap_ctrl == 2 && contour_cone(mrt, newy, ctr.c, tang)))
			return (get_opposite_color(color));
	}
	return (color);
}
