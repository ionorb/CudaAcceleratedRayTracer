#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone_texture.c                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/04/05 07:27:52 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/07 18:42:11 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

static double	get_cone_ang_resol(t_mrt *mrt, t_inter inter, double r_c, \
t_vec cyl_c)
{
	double	arc;
	int		i_res_c;
	int		n_img;
	double	ret;

	ret = r_c;
	arc = r_c / (cyl_c.y * tan(mrt->cone[inter.index].angle / 2));
	i_res_c = (int)((2 * PI) / arc);
	n_img = i_res_c / (mrt->cone[inter.index].option.texture.width - 1);
	if (n_img <= 0)
		ret = (2 * PI) / (mrt->cone[inter.index].option.texture.width - 1);
	else
	{
		if (decimal_part((2 * PI) / n_img) <= 0.5)
			ret = (2 * PI) / (n_img * i_res_c);
		else
			ret = (2 * PI) / ((n_img + 1) * i_res_c);
	}
	return (ret);
}

static double	get_cone_bdy_res(t_mrt *mrt, t_inter inter, double r_c)
{
	int		i_res_h;
	int		n_img;
	double	ret;

	ret = r_c;
	i_res_h = (int)integer_part(mrt->cone[inter.index].height / r_c);
	n_img = i_res_h / (mrt->cone[inter.index].option.texture.height - 1);
	if (n_img <= 0)
		ret = mrt->cone[inter.index].height / \
		(mrt->cone[inter.index].option.texture.height - 1);
	else
	{
		if (decimal_part(mrt->cone[inter.index].height / n_img) <= 0.5)
			ret = mrt->cone[inter.index].height / (n_img * i_res_h);
		else
			ret = mrt->cone[inter.index].height / ((n_img + 1) * i_res_h);
	}
	return (ret);
}

t_rgb	cone_cap_txt_fr_map(t_mrt *mrt, t_inter inter, t_vec c_cr, \
t_c_bump_val values)
{
	int		bump_coor[2];
	t_rgb	color;

	color = inter.color;
	if (values.i_diam > 6)
	{
		bump_coor[0] = (mrt->cone[inter.index].option.texture.height / 2) \
		+ (int)integer_part(c_cr.x / values.res_cap);
		bump_coor[1] = (mrt->cone[inter.index].option.texture.width / 2) \
		+ ((int)integer_part(c_cr.y / values.res_cap));
		pthread_mutex_lock(mrt->mutexs);
		pthread_mutex_unlock(mrt->mutexs);
	}
	pthread_mutex_lock(mrt->mutexs);
	color = convert_to_rgb(mrt->cone[inter.index].option.texture.array \
	[bump_coor[0]][bump_coor[1]]);
	pthread_mutex_unlock(mrt->mutexs);
	return (color);
}

t_rgb	cone_body_txt_fr_map(t_mrt *mrt, t_inter inter, t_vec cyl_c, \
t_c_bump_val values)
{
	int		bump_coor[2];
	double	res_circ;
	double	res_h;
	t_rgb	color;

	color = inter.color;
	res_circ = get_cone_ang_resol(mrt, inter, values.res_cap, cyl_c);
	res_h = get_cone_bdy_res(mrt, inter, values.res_cap);
	bump_coor[1] = (int)integer_part(cyl_c.y / res_circ) % \
	mrt->cone[inter.index].option.texture.width - 1;
	bump_coor[0] = (mrt->cone[inter.index].option.texture.height - 1) \
	- (int)integer_part((cyl_c.z + ((mrt->cone[inter.index].height) / 2)) \
	/ res_h) % (mrt->cone[inter.index].option.texture.height - 1);
	pthread_mutex_lock(mrt->mutexs);
	color = convert_to_rgb(mrt->cone[inter.index].option.texture.array \
	[bump_coor[0]][bump_coor[1]]);
	pthread_mutex_unlock(mrt->mutexs);
	return (color);
}

t_rgb	get_cone_texture(t_mrt *mrt, t_inter inter)
{
	t_mtrx			chg;
	t_vec			new_inter[2];
	t_c_bump_val	values;
	t_rgb			color;

	if (!mrt->cone[inter.index].option.texture_ctrl)
		return (inter.color);
	color = inter.color;
	chg = fill_mtrx(mrt->cone[inter.index].base.n1,
			mrt->cone[inter.index].base.n2,
			mrt->cone[inter.index].base.n3);
	new_inter[0] = vec_rest(inter.inter_coor, mrt->cone[inter.index].pos);
	new_inter[0] = mtrx_by_vec(chg, new_inter[0]);
	new_inter[1] = get_cyl_coor(new_inter[0]);
	values.i_diam = i_min_v(mrt->cone[inter.index].option.texture.height,
			mrt->cone[inter.index].option.texture.width);
	values.res_cap = (2 * mrt->cone[inter.index].height * \
	tan(mrt->cone[inter.index].angle / 2)) / (values.i_diam);
	if (inter.cuad_ctr == 1)
		color = cone_cap_txt_fr_map(mrt, inter, new_inter[0], values);
	if (inter.cuad_ctr == 2)
		color = cone_body_txt_fr_map(mrt, inter, new_inter[1], values);
	return (color);
}
