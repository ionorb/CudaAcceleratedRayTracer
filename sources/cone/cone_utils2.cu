#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone_utils2.c                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/04/07 18:24:18 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/07 18:30:23 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

int	contour_cone(t_mrt *mrt, t_vec *newy, double c, double tang)
{
	t_discr	discr;
	double	radius;

	radius = mrt->cone[mrt->curr_obj.index].height * tang;
	discr = get_cone_disc(newy, tang);
	if ((v_abs(discr.dscr) < 0.0008 * c * radius && !cam_in_cone(mrt, \
	mrt->curr_obj.index, newy[0], tang)) || v_abs((newy[0].z + (c * newy[1].z)) \
	- mrt->cone[mrt->curr_obj.index].height) < 0.05)
		return (1);
	return (0);
}

void	fov_ctr(t_mrt *mrt, int key)
{
	if (mrt->curr_obj.chg_opt == FOV)
	{
		if (key == PLUS && mrt->cone[mrt->curr_obj.index].angle < PI)
			mrt->cone[mrt->curr_obj.index].angle += 0.05;
		if (key == MINUS && mrt->cone[mrt->curr_obj.index].angle > 0.1)
			mrt->cone[mrt->curr_obj.index].angle -= 0.05;
	}
}

t_base	cone_get_tang_base(t_mrt *mrt, t_inter inter, t_vec orig, t_vec cyl_c)
{
	t_base	ret;

	ret = mrt->cone[inter.index].base;
	if (inter.cuad_ctr == 2)
	{
		ret.n3 = orig;
		ret.n1 = fill_coord(cos(cyl_c.y + (PI / 2)), \
		sin(cyl_c.y + (PI / 2)), 0);
		ret.n2 = normalize(cross_prod(ret.n3, ret.n1));
	}
	return (ret);
}

t_vec	cone_bumped(t_mrt *mrt, t_inter inter, t_vec without)
{
	t_mtrx	chg;
	t_vec	new_inter[2];
	t_vec	new_normal;
	t_vec	ret;
	t_base	tang_base;

	ret = without;
	chg = fill_mtrx(mrt->cone[inter.index].base.n1,
			mrt->cone[inter.index].base.n2,
			mrt->cone[inter.index].base.n3);
	new_inter[0] = vec_rest(inter.inter_coor, mrt->cone[inter.index].pos);
	new_inter[0] = mtrx_by_vec(chg, new_inter[0]);
	new_inter[1] = get_cyl_coor(new_inter[0]);
	new_normal = cone_nml_frm_map(mrt, inter, new_inter[0], new_inter[1]);
	tang_base = cone_get_tang_base(mrt, inter, mtrx_by_vec(chg, without), \
	new_inter[1]);
	ret = get_bump_nrml(new_normal, tang_base, mtrx_trsp(chg));
	if (dot_prod(ret, without) < 0)
		ret = normalize(scal_vec(-1, ret));
	return (ret);
}
