#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   save_utils.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/29 22:53:47 by yridgway          #+#    #+#             */
/*   Updated: 2023/04/08 13:28:45 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

void	ft_write_to_file(char *line, int fd)
{
	write(fd, line, ft_strlen(line));
	write(fd, "\n", 1);
}

char	*ft_write_pos(t_vec pos)
{
	char	*line;

	line = ft_strjoin_free(ft_ftoa(pos.x), (","));
	line = ft_strjoin_free(line, ft_ftoa(pos.y));
	line = ft_strjoin_free(line, (","));
	line = ft_strjoin_free(line, ft_ftoa(pos.z));
	return (line);
}

t_vec	ft_unnormalize(t_vec vec)
{
	double	max;

	max = v_abs(vec.x);
	if (v_abs(vec.y) > max)
		max = v_abs(vec.y);
	if (v_abs(vec.z) > max)
		max = v_abs(vec.z);
	vec.x *= 1 / max;
	vec.y *= 1 / max;
	vec.z *= 1 / max;
	return (vec);
}

char	*ft_write_dir(t_vec dir)
{
	char	*line;

	dir = ft_unnormalize(dir);
	line = ft_strjoin_free(ft_ftoa(dir.x), (","));
	line = ft_strjoin_free(line, ft_ftoa(dir.y));
	line = ft_strjoin_free(line, (","));
	line = ft_strjoin_free(line, ft_ftoa(dir.z));
	return (line);
}

char	*ft_write_rgb(t_rgb color)
{
	char	*line;

	line = ft_strjoin_free(ft_itoa(color.r), (","));
	line = ft_strjoin_free(line, ft_itoa(color.g));
	line = ft_strjoin_free(line, (","));
	line = ft_strjoin_free(line, ft_itoa(color.b));
	return (line);
}
