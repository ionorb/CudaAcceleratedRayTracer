#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cell_filling_utils.c                               :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: yoel <yoel@student.42.fr>                  +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/04 01:10:31 by ana               #+#    #+#             */
/*   Updated: 2023/04/02 17:31:01 by yoel             ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

void	ft_check_dots_and_minus(char *str)
{
	int	dot;
	int	minus;
	int	i;

	i = 0;
	dot = 0;
	minus = 0;
	while (str[i])
	{
		if (str[i] == '.')
			dot++;
		if (str[i] == '-')
			minus++;
		if (dot > 1)
			ft_error("Multiple dots(.) in value", str, NULL);
		if (minus > 1)
			ft_error("Multiple minuses(-) in value", str, NULL);
		if (i > 0 && str[i] == '-' && str[i - 1] != ' ')
			ft_error("Minus(-) in value", str, NULL);
		if (str[i] == '.' && !str[i + 1])
			ft_error("Dot(.) in value", str, NULL);
		i++;
	}
}

int	check_valid_number(char *str)
{
	int	i;
	int	count;

	count = 0;
	i = 0;
	ft_check_dots_and_minus(str);
	while (str[i] == '0')
		i++;
	while (str[i] >= '0' && str[i] <= '9')
	{
		count++;
		i++;
		if (count > 10)
			return (ft_error(INT_RANGE, str, NULL), 0);
	}
	if (str[i++] == '.')
	{
		count = 0;
		while (str[i + count] >= '0' && str[i + count] <= '9' && count < 15)
			count++;
	}
	if (count >= 15)
		return (ft_error(DOUBLE_RANGE, str, NULL), 0);
	return (1);
}

void	valid_nums(char **line)
{
	while (line && *line)
		check_valid_number(*line++);
}

int	check_for_chars(char *str, char *cell)
{
	int	i;

	i = 0;
	while (cell && cell[i])
	{
		if ((cell[i] == '.' && cell[i + 1] == '-') || \
		(cell[i] == '-' && cell[i + 1] == ',') || \
		(cell[i] == '-' && cell[i + 1] == '.') || \
		(cell[i] == ',' && cell[i + 1] == '.') || \
		(cell[i] == '.' && cell[i + 1] == ','))
			return (1);
		if (!ft_strchr(str, cell[i]))
			return (1);
		i++;
	}
	if (!cell || !cell[0] || cell[0] == ',' || cell[i - 1] == ',' \
	|| cell[0] == '.' || cell[i - 1] == '.')
		return (1);
	return (0);
}

int	out_of_range(double num, double min, double max)
{
	if (num < min || num > max)
		return (1);
	return (0);
}
