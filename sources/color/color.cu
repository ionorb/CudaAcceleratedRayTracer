#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   color.c                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: yridgway <yridgway@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/01 01:47:46 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/07 13:44:35 by yridgway         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_vec	get_normal_at_point(t_mrt *mrt, t_inter inter)
{
	t_vec	ret;

	ret = fill_coord(0, 0, 0);
	if (inter.type == PLANE)
		ret = get_normal_plane(mrt, inter);
	else if (inter.type == SPHERE)
		ret = get_normal_sphere(mrt, inter);
	else if (inter.type == CYLINDER)
		ret = get_normal_cylinder(mrt, inter);
	else if (inter.type == TRIANGLE)
		ret = get_normal_triangle(mrt, inter);
	else if (inter.type == CONE)
		ret = get_normal_cone(mrt, inter);
	return (ret);
}

// t_rgb	get_object_color(t_mrt *mrt, t_inter inter)
// {
// }

t_rgb	get_radiance(t_mrt *mrt, t_inter *ctr, t_light light)
{
	t_vec	to_light;
	t_inter	linter;
	t_rgb	diffuse;
	t_rgb	specular;

	diffuse = ft_make_rgb(0, 0, 0);
	specular = ft_make_rgb(0, 0, 0);
	to_light = vec_rest(light.pos, ctr->inter_coor);
	linter = check_shaddow(mrt, ctr, normalize(to_light), vect_norm(to_light));
	if ((linter.dist < 0 || linter.dist > vect_norm(to_light)))
	{
		if (ctr->option.mirror < 1.0)
			diffuse = get_diffuse(ctr, to_light, light);
		if (ctr->option.specular[0] > 0)
			specular = get_specular(ctr, mrt->cam.pos, to_light, light);
	}
	diffuse = mult_color(diffuse, 1 - ctr->option.mirror);
	specular = mult_color(specular, 1 - ctr->option.mirror);
	return (add_color(diffuse, specular));
}

t_rgb	apply_lighting(t_mrt *mrt, t_inter *ctr, t_vec dir, t_rgb color)
{
	int		i;
	t_rgb	reflection;

	i = -1;
	if (ctr->dist == -1)
		return (ft_make_rgb(0, 0, 0));
	// reflection = ft_make_rgb(0, 0, 0);
	while (++i < mrt->obj_count[LIGHT])
		color = add_color(color, \
		get_radiance(mrt, ctr, mrt->light[i]));
	// if (ctr->option.mirror > 0 && mrt->bounce < 40)
	// 	reflection = mult_color(get_reflection(mrt, ctr, dir), \
	// 	ctr->option.mirror);
	// color = add_color(color, reflection);
	return (add_color(color, get_ambient(ctr->color, mrt->amblight, 1)));
}

int	get_pixel_color(t_mrt *mrt, int x, int y)
{
	t_inter	inter;
	t_rgb	color;
	t_vec	dir;

	dir = normalize(vec_rest(screen_pxl_by_indx(mrt, \
	&mrt->cam, x + 1, y + 1), mrt->cam.pos));
	color = ft_make_rgb(0, 0, 0);
	inter = check_intersections(mrt, mrt->cam.pos, dir);
	if (inter.dist != -1)
	{
		inter.norm = get_normal_at_point(mrt, inter);
		color = apply_lighting(mrt, &inter, dir, color);
		color = chosen_obj(mrt, x, y, color);
	}
	mrt->bounce = 0;
	color = normalize_color(color);
	color = show_light_sources(mrt, color, dir);
	return ((int)color.r << 16 | (int)color.g << 8 | (int)color.b);
}
