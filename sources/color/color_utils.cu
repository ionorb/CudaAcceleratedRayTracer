#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   color_utils.c                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/03 18:55:19 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/05 06:45:50 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

// double	get_angle_between(t_vec v1, t_vec v2)
// {
// 	double	angle;

// 	if (vect_norm(v1) == 0 || vect_norm(v2) == 0)
// 		return (0);
// 	angle = acos(dot_prod(v1, v2) / (vect_norm(v1) * vect_norm(v2)));
// 	return (angle);
// }

t_rgb	ft_make_rgb_ratio(t_rgb color)
{
	color.r = color.r / 255;
	color.g = color.g / 255;
	color.b = color.b / 255;
	return (color);
}

t_rgb	normalize_color(t_rgb color)
{
	double	max;

	max = color.r;
	if (color.g > max)
		max = color.g;
	if (color.b > max)
		max = color.b;
	if (max > 255)
	{
		color.r *= 255 / max;
		color.g *= 255 / max;
		color.b *= 255 / max;
	}
	return (color);
}

t_rgb	ft_make_rgb(int r, int g, int b)
{
	t_rgb	color;

	color.r = r;
	color.g = g;
	color.b = b;
	return (color);
}

t_rgb	show_light_sources(t_mrt *mrt, t_rgb color, t_vec dir)
{
	int		i;
	t_vec	cam_to_light;
	t_inter	linter;
	t_inter	ctr;
	t_discr	dscr;

	i = -1;
	ctr.inter_coor = mrt->cam.pos;
	ctr.norm = mrt->cam.dir;
	while (++i < mrt->obj_count[LIGHT])
	{
		cam_to_light = vec_rest(mrt->light[i].pos, mrt->cam.pos);
		linter = check_shaddow(mrt, &ctr, normalize(cam_to_light), \
		vect_norm(cam_to_light));
		if ((linter.dist < 0 || linter.dist > vect_norm(cam_to_light)))
		{
			dscr = get_sph_dscr(vec_rest(mrt->cam.pos, \
			mrt->light[i].pos), dir, int_pow(0.2, 2));
			if (dscr.dscr >= 0 && solve_quad(&dscr) > 0)
				color = mrt->light[i].color;
		}
	}
	return (color);
}
