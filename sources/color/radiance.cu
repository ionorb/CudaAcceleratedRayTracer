#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   radiance.c                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: yridgway <yridgway@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/15 01:29:54 by yoel              #+#    #+#             */
/*   Updated: 2023/04/07 13:44:10 by yridgway         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_rgb	get_ambient(t_rgb ctr, t_light amb, double mirror)
{
	t_rgb	ratio;
	t_rgb	color;

	ratio = ft_make_rgb_ratio(ctr);
	color.r = amb.color.r * amb.ratio * ratio.r * mirror;
	color.g = amb.color.g * amb.ratio * ratio.g * mirror;
	color.b = amb.color.b * amb.ratio * ratio.b * mirror;
	return (color);
}

t_rgb	get_reflection(t_mrt *mrt, t_inter *ctr, t_vec dir)
{
	t_vec	refl_dir;
	t_vec	point;
	t_inter	refl_inter;
	t_rgb	color;

	mrt->bounce++;
	refl_dir = vec_sum(dir, scal_vec(-2 * dot_prod(dir, ctr->norm), \
	ctr->norm));
	point = vec_sum(ctr->inter_coor, scal_vec(0.0000001, ctr->norm));
	refl_inter = check_intersections(mrt, point, refl_dir);
	refl_inter.norm = get_normal_at_point(mrt, refl_inter);
	color = apply_lighting(mrt, &refl_inter, refl_dir, ft_make_rgb(0, 0, 0));
	return (color);
}

t_rgb	get_diffuse(t_inter *ctr, t_vec to_light, t_light light)
{
	double	angle;
	t_rgb	ratio;
	t_rgb	color;

	angle = dot_prod(normalize(ctr->norm), normalize(to_light));
	if (angle < 0)
		angle = 0;
	ratio = ft_make_rgb_ratio(ctr->color);
	color.r = light.color.r * light.ratio * ratio.r * angle;
	color.g = light.color.g * light.ratio * ratio.g * angle;
	color.b = light.color.b * light.ratio * ratio.b * angle;
	return (color);
}

t_rgb	get_specular(t_inter *ctr, t_vec pos, t_vec to_light, t_light light)
{
	double	ratio;
	t_rgb	color;
	t_vec	h;
	int		exponent;
	double	intensity;

	intensity = ctr->option.specular[0];
	exponent = (int)ctr->option.specular[1];
	h = normalize(scal_vec(1 / vect_norm(normalize(vec_sum(normalize(to_light), \
	normalize(vec_rest(pos, ctr->inter_coor))))), \
	normalize(vec_sum(normalize(to_light), \
	normalize(vec_rest(pos, ctr->inter_coor))))));
	ratio = \
	intensity * int_pow(dot_prod(normalize(ctr->norm), h), exponent * 10);
	color.r = light.color.r * light.ratio * ratio;
	color.g = light.color.g * light.ratio * ratio;
	color.b = light.color.b * light.ratio * ratio;
	return (color);
}
