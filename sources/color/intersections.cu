#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   intersections.c                                    :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/04/01 20:17:11 by yridgway          #+#    #+#             */
/*   Updated: 2023/04/05 01:59:53 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_inter	check_intersections(t_mrt *mrt, t_vec point, t_vec dir)
{
	t_inter	ret;

	ret.type = UNDEFINED;
	ret.index = 0;
	ret.dist = -1;
	check_planes(mrt, &ret, point, dir);
	check_spheres(mrt, &ret, point, dir);
	check_cylinders(mrt, &ret, point, dir);
	check_triangles(mrt, &ret, point, dir);
	check_cones(mrt, &ret, point, dir);
	return (ret);
}

t_inter	check_shaddow(t_mrt *mrt, t_inter *ctr, t_vec dir, double len)
{
	t_inter	ret;
	t_vec	point;

	point = vec_sum(ctr->inter_coor, scal_vec(0.0000001, ctr->norm));
	ret.type = UNDEFINED;
	ret.index = 0;
	ret.dist = -1;
	check_planes(mrt, &ret, point, dir);
	if (ret.dist != -1 && ret.dist < len)
		return (ret);
	check_spheres(mrt, &ret, point, dir);
	if (ret.dist != -1 && ret.dist < len)
		return (ret);
	check_cylinders(mrt, &ret, point, dir);
	if (ret.dist != -1 && ret.dist < len)
		return (ret);
	check_cones(mrt, &ret, point, dir);
	if (ret.dist != -1 && ret.dist < len)
		return (ret);
	check_triangles(mrt, &ret, point, dir);
	return (ret);
}
