/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   init.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/02/13 20:51:49 by yridgway          #+#    #+#             */
/*   Updated: 2023/04/07 18:18:54 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

// int	ft_init_mlx(t_mrt *mrt)
// {
// 	if (!mrt->save)
// 	{
// 		mrt->win = mlx_new_window(mrt->mlx, mrt->ix, mrt->iy, "MiniRT");
// 		if (!mrt->win)
// 			return (mlx_destroy_display(mrt->mlx), free(mrt->mlx), \
// 			ft_error("Problem initializing minilibx", NULL, NULL), 1);
// 	}
// 	mrt->img = mlx_new_image(mrt->mlx, mrt->ix, mrt->iy);
// 	if (!mrt->img)
// 		return (mlx_destroy_window(mrt->mlx, mrt->win), \
// 		mlx_destroy_display(mrt->mlx), free(mrt->mlx), \
// 		ft_error("Problem initializing minilibx", NULL, NULL), 1);
// 	mrt->addr = mlx_get_data_addr(mrt->img, &mrt->bpp, &mrt->sizel, &mrt->endi);
// 	if (!mrt->addr)
// 		return (ft_error("Problem initializing minilibx", NULL, NULL), 1);
// 	ft_memory(mrt, SAVE_MLX);
// 	return (0);
// }

int	valid_rt_file(char *file, int fd)
{
	int	size;

	if (!file || !*file)
		return (0);
	size = ft_strlen(file);
	if (size < 3)
		return (0);
	if (file[size - 1] != 't' || file[size - 2] != 'r' || file[size - 3] != '.')
		return (0);
	if (read(fd, NULL, 0) < 0)
		ft_error("Failed to read file", file, strerror(errno));
	return (1);
}

void	ft_reinit(t_mrt *mrt)
{
	int	i;

	i = 0;
	ft_free(mrt->sphere);
	mrt->sphere = NULL;
	ft_free(mrt->plane);
	mrt->plane = NULL;
	ft_free(mrt->cylinder);
	mrt->cylinder = NULL;
	ft_free(mrt->cone);
	mrt->cone = NULL;
	ft_free(mrt->light);
	mrt->light = NULL;
	ft_free(mrt->triangle);
	mrt->triangle = NULL;
	while (i < mrt->num_objs)
		mrt->obj_count[i++] = 0;
	mrt->curr_obj = define_curr_obj(CAMERA, 0);
	ft_parse(mrt);
}

int	init_minirt(t_mrt *mrt, char *av1, int ac)
{
	(void)ac;
	// if (mrt->save)
		// ft_set_mrt(mrt, av[1], ft_atoi(av[3]), ft_atoi(av[4]));
	// else
	ft_set_mrt(mrt, av1, IX, IY);
	// mrt->mlx = mlx_init();
	// ft_memory(mrt, SAVE_MLX);
	// if (!mrt->mlx)
		// return (ft_error("Problem initializing minilibx", NULL, NULL), 0);
	ft_parse(mrt);
	CUDA_CALL(hipMallocManaged(&mrt->addr, sizeof(char) * IX * IY * 4));
	// if (ft_init_mlx(mrt))
	// 	return (printf("Problem initializing minilibx\n"), 1);
	return (0);
}
