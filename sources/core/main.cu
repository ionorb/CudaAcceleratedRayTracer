/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   main.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: yoel <yoel@student.42.fr>                  +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/05 21:18:58 by ana               #+#    #+#             */
/*   Updated: 2023/09/30 18:00:52 by yoel             ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "minirt.h"

int	end_mrt(int i, void *mrt)
{
	(void)mrt;
	(void)i;
	printf("%s\n", CLEAN_EXIT);
	ft_quit(EXIT_OK);
	return (0);
}

// int	ft_controls(t_mrt *mrt)
// {
// 	mlx_hook(mrt->win, 2, 1, key_press, mrt);
// 	mlx_hook(mrt->win, 17, 0, end_mrt, mrt);
// 	mlx_mouse_hook(mrt->win, &mouse_press, mrt);
// 	return (0);
// }

void	render_scene(t_mrt *mrt)
{
	set_all_cam_values(&mrt->cam, mrt->ix);
	if (mrt->first)
		write(1, "calculating pixel values...\n", 29);
	pixel_calcul(mrt);
	write_to_ppm(mrt);
	// if (!mrt->save)
	// {
	// 	mlx_clear_window(mrt->mlx, mrt->win);
	// 	mlx_put_image_to_window(mrt->mlx, mrt->win, mrt->img, 0, 0);
	// 	display_strings(mrt);
	// }
	if (mrt->first)
		mrt->first = 0;
}

int	ft_save_parsing(char **av)
{
	if (ft_strcmp_1(av[2], "--save"))
		return (printf("Usage: ./miniRT <scene.rt> --save\n"), 1);
	if (ft_atoi(av[3]) < 1 || ft_atoi(av[4]) < 1 || ft_strlen(av[3]) > 6 \
	|| ft_strlen(av[4]) > 6 || ft_atoi(av[3]) > 10000 \
	|| ft_atoi(av[4]) > 10000)
		return (printf("Please enter a save resolution between 20 and 10000\n"), \
		1);
	return (0);
}

int	main(int ac, char **av)
{
	t_mrt	*mrt;

	CUDA_CALL(hipMallocManaged(&mrt, sizeof(t_mrt)));
	mrt->save = 1;
	if (ac != 2 && ac != 5)
		return (printf("Usage: ./miniRT <scene.rt>\n"), 1);
	if (ac == 5)
	{
		mrt->save = 1;
		if (ft_save_parsing(av))
			return (1);
	}
	write(1, "initializing minirt... ", 23);
	if (init_minirt(mrt, av[1], ac))
		return (1);
	write(1, "done\n", 5);
	mrt->first = 1;
	return (render_scene(mrt), ft_quit(EXIT_OK), 0);
	// ft_controls(&mrt);
	// mlx_loop(mrt.mlx);
}
