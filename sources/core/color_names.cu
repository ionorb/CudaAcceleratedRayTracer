#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   color_names.c                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/24 18:13:47 by yridgway          #+#    #+#             */
/*   Updated: 2023/04/07 19:49:04 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_rgb	get_ratio_rgb(t_rgb color)
{
	double	max;

	max = color.r;
	if (color.g > max)
		max = color.g;
	if (color.b > max)
		max = color.b;
	if (max > 1)
	{
		color.r *= 1 / max;
		color.g *= 1 / max;
		color.b *= 1 / max;
	}
	return (color);
}

char	*white_grey_black(t_rgb color)
{
	int	tmp;

	tmp = color.r + color.g + color.b;
	if (tmp > 150 && tmp < 678)
		return (("grey"));
	if (tmp > 678)
		return (("white"));
	return (("black"));
}

char	*ft_get_color_str(t_rgb color)
{
	double	diff;
	char	**str;
	char	*ret;
	int		i;

	i = 0;
	ret = NULL;
	diff = 255 * 3;
	str = ft_split(COLORS, '\n');
	if (color.r == color.g && color.g == color.b)
		return (white_grey_black(color));
	color = get_ratio_rgb(color);
	while (str[i])
	{
		if (v_abs(color.r - ft_atof(str[i])) + v_abs(color.g - \
		ft_atof(str[i] + 4)) + v_abs(color.b - ft_atof(str[i] + 8)) < diff)
		{
			diff = v_abs(color.r - ft_atof(str[i])) + v_abs(color.g - \
			ft_atof(str[i] + 4)) + v_abs(color.b - ft_atof(str[i] + 8));
			ret = ft_strdup(str[i] + 13);
		}
		i++;
	}
	return (ret);
}

t_rgb	ft_get_obj_color(t_mrt *mrt, int type, int index)
{
	t_rgb	color;

	color.r = 0;
	color.g = 0;
	color.b = 0;
	if (type == SPHERE)
		color = mrt->sphere[index].color;
	if (type == PLANE)
		color = mrt->plane[index].color;
	if (type == CYLINDER)
		color = mrt->cylinder[index].color;
	if (type == LIGHT)
		color = mrt->light[index].color;
	return (color);
}
