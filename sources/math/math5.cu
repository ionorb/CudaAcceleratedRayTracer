#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   math5.c                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/05 21:20:39 by ana               #+#    #+#             */
/*   Updated: 2023/04/05 06:45:25 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_vec	cross_prod(t_vec v1, t_vec v2)
{
	t_vec	ret;

	ret.x = v1.y * v2.z - v1.z * v2.y;
	ret.y = v1.z * v2.x - v1.x * v2.z;
	ret.z = v1.x * v2.y - v1.y * v2.x;
	return (ret);
}

double	perp_to_plane(t_vec point, t_vec plane_point, t_vec plane_norm)
{
	double		distance;
	t_vec		plane_to_point;

	plane_to_point = vec_rest(point, plane_point);
	distance = v_abs(dot_prod(plane_to_point, plane_norm)
			/ vect_norm(plane_norm));
	return (distance);
}

t_mtrx	fill_mtrx(t_vec v1, t_vec v2, t_vec v3)
{
	t_mtrx	ret;

	ret.r1 = v1;
	ret.r2 = v2;
	ret.r3 = v3;
	return (ret);
}

t_mtrx	init_base_mtrx(t_base *base)
{
	t_mtrx	ret;
	t_vec	curr;

	curr = fill_coord(base->n1.x, base->n2.x, base->n3.x);
	ret.r1 = curr;
	curr = fill_coord(base->n1.y, base->n2.y, base->n3.y);
	ret.r2 = curr;
	curr = fill_coord(base->n1.z, base->n2.z, base->n3.z);
	ret.r3 = curr;
	return (ret);
}

double	solve_quad(t_discr *info)
{
	double	op1;
	double	op2;
	double	ret;

	if (info->dscr < 0.00001)
		return (-info->b / (2 * info->a));
	op1 = (-info->b + sqrt(info->dscr)) / (2 * info->a);
	op2 = (-info->b - sqrt(info->dscr)) / (2 * info->a);
	ret = min_v(op1, op2);
	if (ret < 0)
		return (max_v(op1, op2));
	return (ret);
}
