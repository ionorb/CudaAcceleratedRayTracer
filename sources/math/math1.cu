#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   math1.c                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: ana <ana@student.42.fr>                    +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/02/14 21:53:22 by gamoreno          #+#    #+#             */
/*   Updated: 2023/03/05 21:12:31 by ana              ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

/*if ctrl == 0, it passes radians to degrees, else it passes degrees
to radians */
double	rad_and_deg(double angle, int ctrl)
{
	if (ctrl == 0)
		return ((angle * 180) / PI);
	else
		return ((angle * PI) / 180);
}

double	int_pow(double basis, int exp)
{
	int		i;
	double	res;

	res = 1;
	i = 0;
	while (i < exp)
	{
		res *= basis;
		i++;
	}
	return (res);
}

t_vec	fill_coord(double c1, double c2, double c3)
{
	t_vec	res;

	res.x = c1;
	res.y = c2;
	res.z = c3;
	return (res);
}

double	vect_norm(t_vec v)
{
	return (sqrt(norm_raised_2(v)));
}

t_vec	normalize(t_vec v)
{
	t_vec	ret;
	double	norm;

	norm = vect_norm(v);
	ret.x = v.x / norm;
	ret.y = v.y / norm;
	ret.z = v.z / norm;
	return (ret);
}
