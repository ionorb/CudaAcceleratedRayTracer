#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   math7.c                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/13 20:54:16 by gamoreno          #+#    #+#             */
/*   Updated: 2023/03/23 06:47:23 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_mtrx	define_rot_mtrx(t_vec rot_axis, double sin, double cos)
{
	t_mtrx	ret;

	ret.r1 = fill_coord(cos + (int_pow(rot_axis.x, 2) * (1 - cos)),
			(rot_axis.x * rot_axis.y * (1 - cos)) - (rot_axis.z * sin),
			(rot_axis.x * rot_axis.z * (1 - cos)) + (rot_axis.y * sin));
	ret.r2 = fill_coord((rot_axis.x * rot_axis.y * (1 - cos))
			+ (rot_axis.z * sin), cos + (int_pow(rot_axis.y, 2)
				* (1 - cos)), (rot_axis.y * rot_axis.z * (1 - cos))
			- (rot_axis.x * sin));
	ret.r3 = fill_coord((rot_axis.x * rot_axis.z * (1 - cos))
			- (rot_axis.y * sin), (rot_axis.y * rot_axis.z * (1 - cos))
			+ (rot_axis.x * sin), cos + (int_pow(rot_axis.z, 2)
				* (1 - cos)));
	return (ret);
}

double	integer_part(double n)
{
	double	integer_part;
	double	decimal_part;

	decimal_part = modf(n, &integer_part);
	(void)decimal_part;
	return (integer_part);
}

double	get_azimuth(t_vec orig)
{
	double	ret;

	ret = 0.0;
	if (orig.z > 0)
		ret = atan(sqrt(int_pow(orig.x, 2) + int_pow(orig.y, 2)) / orig.z);
	if (orig.z == 0)
		ret = PI / 2;
	if (orig.z < 0)
		ret = PI + atan(sqrt(int_pow(orig.x, 2)
					+ int_pow(orig.y, 2)) / orig.z);
	return (ret);
}

t_vec	get_spheric_coord(t_vec orig)
{
	t_vec	ret;

	ret.x = vect_norm(orig);
	ret.y = get_azimuth(orig);
	if (orig.x > 0)
	{
		if (orig.y > 0)
			ret.z = atan(orig.y / orig.x);
		if (orig.y < 0)
			ret.z = (2 * PI) + atan(orig.y / orig.x);
	}
	if (orig.x == 0)
	{
		if (orig.y > 0)
			ret.z = PI / 2;
		if (orig.y < 0)
			ret.z = -PI / 2;
	}
	if (orig.x < 0)
		ret.z = PI + atan(orig.y / orig.x);
	return (ret);
}

t_vec	get_cyl_coor(t_vec orig)
{
	t_vec	ret;

	ret.x = sqrt(int_pow(orig.x, 2) + int_pow(orig.y, 2));
	if (orig.x != 0)
	{
		ret.y = atan(orig.y / orig.x);
		if (ret.y < 0)
			ret.y += PI;
	}
	else
	{
		if (orig.y > 0)
			ret.y = PI;
		if (orig.y < 0)
			ret.y = -PI;
	}
	ret.z = orig.z;
	return (ret);
}
