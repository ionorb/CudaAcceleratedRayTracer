#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   math6.c                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: yridgway <yridgway@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/13 20:54:16 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/01 20:27:39 by yridgway         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_base	first_rotation(t_vec dir, t_base can)
{
	double	sin_an;
	double	cos_an;
	t_vec	curr;
	t_base	ret;
	t_mtrx	chng_base;

	curr = vec_rest(dir, scal_vec(dir.y, can.n2));
	cos_an = dot_prod(can.n3, curr) / vect_norm(curr);
	sin_an = sqrt(1 - int_pow(cos_an, 2));
	if (dir.x < 0)
		sin_an *= -1;
	chng_base.r1 = fill_coord(cos_an, 0, sin_an);
	chng_base.r2 = fill_coord(0, 1, 0);
	chng_base.r3 = fill_coord(-sin_an, 0, cos_an);
	ret.n1 = mtrx_by_vec(chng_base, can.n1);
	ret.n2 = mtrx_by_vec(chng_base, can.n2);
	ret.n3 = mtrx_by_vec(chng_base, can.n3);
	return (ret);
}

t_base	second_rotation(t_vec dir, t_base ret)
{
	double	sin_an;
	double	cos_an;
	t_base	retu;
	t_mtrx	chng_base;

	cos_an = dot_prod(dir, ret.n3);
	sin_an = -sqrt(1 - int_pow(cos_an, 2));
	if (dir.y < 0)
		sin_an *= -1;
	chng_base.r1 = fill_coord(cos_an + (int_pow(ret.n1.x, 2) * (1 - cos_an)),
			(ret.n1.x * ret.n1.y * (1 - cos_an)) - (ret.n1.z * sin_an),
			(ret.n1.x * ret.n1.z * (1 - cos_an)) + (ret.n1.y * sin_an));
	chng_base.r2 = fill_coord((ret.n1.x * ret.n1.y * (1 - cos_an))
			+ (ret.n1.z * sin_an), cos_an + (int_pow(ret.n1.y, 2)
				* (1 - cos_an)), (ret.n1.y * ret.n1.z * (1 - cos_an))
			- (ret.n1.x * sin_an));
	chng_base.r3 = fill_coord((ret.n1.x * ret.n1.z * (1 - cos_an))
			- (ret.n1.y * sin_an), (ret.n1.y * ret.n1.z * (1 - cos_an))
			+ (ret.n1.x * sin_an), cos_an + (int_pow(ret.n1.z, 2)
				* (1 - cos_an)));
	retu.bs_orig = fill_coord(0, 0, 0);
	retu.n1 = mtrx_by_vec(chng_base, ret.n1);
	retu.n2 = mtrx_by_vec(chng_base, ret.n2);
	retu.n3 = mtrx_by_vec(chng_base, ret.n3);
	return (retu);
}

t_base	general_rotation(t_base base, int ctrl, double rad)
{
	double	sin_an;
	double	cos_an;
	t_vec	rot_axis;
	t_base	ret;
	t_mtrx	chng_base;

	if (ctrl == 1)
		rot_axis = base.n1;
	else if (ctrl == 2)
		rot_axis = base.n2;
	else if (ctrl == 3)
		rot_axis = base.n3;
	else
		return (base);
	cos_an = cos(rad);
	sin_an = sin(rad);
	chng_base = define_rot_mtrx(rot_axis, sin_an, cos_an);
	ret.n1 = mtrx_by_vec(chng_base, base.n1);
	ret.n2 = mtrx_by_vec(chng_base, base.n2);
	ret.n3 = mtrx_by_vec(chng_base, base.n3);
	return (ret);
}

t_base	get_obj_base(t_vec	dir)
{
	t_base	ret;
	t_base	can;

	can.n1 = fill_coord(1, 0, 0);
	can.n2 = fill_coord(0, 1, 0);
	can.n3 = fill_coord(0, 0, 1);
	if (v_abs(dir.x) < 0.00001 && v_abs(dir.z) < 0.00001)
	{
		ret.n1 = can.n3;
		ret.n2 = can.n1;
		ret.n3 = can.n2;
		return (ret);
	}
	ret = first_rotation(dir, can);
	ret = second_rotation(dir, ret);
	return (ret);
}

double	decimal_part(double n)
{
	double	integer_part;
	double	decimal_part;

	decimal_part = modf(n, &integer_part);
	return (decimal_part);
}
