#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   math.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: yridgway <yridgway@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/02/14 21:53:22 by gamoreno          #+#    #+#             */
/*   Updated: 2023/03/26 23:40:28 by yridgway         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

double	int_pow(double basis, int exp)
{
	int		i;
	double	res;

	res = 1;
	i = 0;
	while (i < exp)
	{
		res *= basis;
		i++;
	}
	return (res);
}

double	vect_norm(t_vec v)
{
	double	sum_sqr;

	sum_sqr = int_pow(v.x, 2) + int_pow(v.y, 2) + int_pow(v.z, 2);
	return (sqrt(sum_sqr));
}

t_vec	normalize(t_vec v)
{
	t_vec	ret;
	double	norm;

	norm = vect_norm(v);
	ret.x = v.x / norm;
	ret.y = v.y / norm;
	ret.z = v.z / norm;
	return (ret);
}

t_vec	vector_sum(t_vec v1, t_vec v2)
{
	t_vec	res;

	res.x = v1.x + v2.x;
	res.y = v1.y + v2.y;
	res.z = v1.z + v2.z;
	return (res);
}
