/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   paint_utils.c                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: yridgway <yridgway@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/04/01 21:16:00 by yridgway          #+#    #+#             */
/*   Updated: 2023/04/05 04:09:44 by yridgway         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

// void	ft_percentage_bar(t_mrt *mrt)
// {
// 	double	percent;

// 	if (mrt->first)
// 	{
// 		pthread_mutex_lock(mrt->mutexs);
// 		percent = (double)*mrt->percent / (mrt->ix * THREADS) * 100;
// 		printf("\r[%.0f%%]", percent);
// 		(*mrt->percent)++;
// 		pthread_mutex_unlock(mrt->mutexs);
// 	}
// }

void	ft_copy_mem(t_mrt *mrt, t_mrt *dat)
{
	dat->light = (t_light*)ft_memcpy(mrt->light, mrt->obj_count[LIGHT] * sizeof(t_light));
	if (mrt->sphere)
		dat->sphere = \
		(t_sphere*)ft_memcpy(mrt->sphere, mrt->obj_count[SPHERE] * sizeof(t_sphere));
	if (mrt->plane)
		dat->plane = \
		(t_plane*)ft_memcpy(mrt->plane, mrt->obj_count[PLANE] * sizeof(t_plane));
	if (mrt->cylinder)
		dat->cylinder = \
		(t_cylinder*)ft_memcpy(mrt->cylinder, mrt->obj_count[CYLINDER] * sizeof(t_cylinder));
	if (mrt->cone)
		dat->cone = (t_cone*)ft_memcpy(mrt->cone, mrt->obj_count[CONE] * sizeof(t_cone));
	if (mrt->triangle)
		dat->triangle = \
		(t_triangle*)ft_memcpy(mrt->triangle, mrt->obj_count[TRIANGLE] * sizeof(t_triangle));
	dat->curr_obj = mrt->curr_obj;
	dat->scene_path = mrt->scene_path;
	dat->obj_count = \
	(int*)ft_memcpy(mrt->obj_count, mrt->num_objs * sizeof(int));
}

t_mrt	*ft_copy_mrt(t_mrt *mrt)
{
	int		i;
	t_mrt	*dat;

	i = -1;
	hipMallocManaged(dat, sizeof(t_mrt));
	ft_set_mrt(dat, "unset", 0, 0);
	dat->save = mrt->save;
	dat->addr = mrt->addr;
	dat->bpp = mrt->bpp;
	dat->endi = mrt->endi;
	dat->sizel = mrt->sizel;
	dat->mutexs = &mrt->mutex;
	dat->first = mrt->first;
	dat->num_objs = mrt->num_objs;
	dat->bounce = mrt->bounce;
	dat->ix = mrt->ix;
	dat->iy = mrt->iy;
	dat->amblight = mrt->amblight;
	dat->cam = mrt->cam;
	dat->percent = &mrt->i;
	ft_copy_mem(mrt, dat);
	return (dat);
}

void	ft_free_mrt(t_mrt *mrt, int num)
{
	int	i;

	i = -1;
	while (++i < num)
	{
		ft_free(mrt[i].light);
		ft_free(mrt[i].sphere);
		ft_free(mrt[i].plane);
		ft_free(mrt[i].cylinder);
		ft_free(mrt[i].cone);
		ft_free(mrt[i].triangle);
		ft_free(mrt[i].obj_count);
		ft_free(mrt[i].scene_path);
	}
	ft_free(mrt);
}
