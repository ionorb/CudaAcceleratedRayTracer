#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cam.c                                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/02/15 00:02:45 by gamoreno          #+#    #+#             */
/*   Updated: 2023/02/27 15:48 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

void	set_all_cam_values(t_cam *cam, int ix)
{
	double	aux_norm;
	double	sqr_sum1;
	double	sqr_sum2;

	cam->s_half_screen = tan(rad_and_deg(cam->fov / 2, 1));
	cam->step = cam->s_half_screen / ix;
	cam->screen_base.bs_orig = vec_sum(cam->pos, cam->dir);
	if (v_abs(cam->dir.x) < 0.00001 && v_abs(cam->dir.y) < 0.0001)
	{
		cam->screen_base.n1 = fill_coord(1, 0, 0);
		cam->screen_base.n2 = fill_coord(0, 1, 0);
		cam->screen_base.n3 = fill_coord(0, 0, cam->dir.z);
		return ;
	}
	sqr_sum1 = int_pow(cam->dir.x, 2) + int_pow(cam->dir.y, 2);
	aux_norm = sqrt(sqr_sum1);
	cam->screen_base.n1 = fill_coord(cam->dir.y / aux_norm,
			-cam->dir.x / aux_norm, 0);
	sqr_sum2 = int_pow(cam->dir.x * cam->dir.z, 2)
		+ int_pow(cam->dir.y * cam->dir.z, 2) + int_pow(sqr_sum1, 2);
	aux_norm = sqrt(sqr_sum2);
	cam->screen_base.n2 = fill_coord((cam->dir.x * cam->dir.z) / aux_norm,
			(cam->dir.y * cam->dir.z) / aux_norm, -sqr_sum1 / aux_norm);
	cam->screen_base.n3 = fill_coord(cam->dir.x, cam->dir.y, cam->dir.z);
}

t_vec	screen_pxl_by_indx(t_mrt *mrt, t_cam *cam, int i, int j)
{
	t_vec	res;
	double	scal_i;
	double	scal_j;

	scal_i = -cam->s_half_screen + (((2 * i) - 1) * cam->step);
	scal_j = ((-(double)mrt->iy / (double)mrt->ix) * cam->s_half_screen)
		+ (((2 * j) - 1) * cam->step);
	res = vec_sum(cam->screen_base.bs_orig,
			vec_sum(scal_vec(scal_i, cam->screen_base.n1),
				scal_vec(scal_j, cam->screen_base.n2)));
	return (res);
}
