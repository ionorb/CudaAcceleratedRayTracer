#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   paint.c                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/02/21 22:24:35 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/07 18:23:52 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

void	my_mlx_pixel_put(t_mrt *mrt, int x, int y, int color)
{
	char	*dst;

	// if (!mrt->save && x < BORDER)
		// color = diminish_color(color, 0.3);
	dst = mrt->addr + (y * mrt->sizel + x * (mrt->bpp / 8));
	*(unsigned int *)dst = color;
}

void __global__ ft_paint_device(t_mrt *mrt, int size)
{
	// int		j, i = 0;
	int		color;

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	(void)index;
	(void)stride;

	int x, y = 0;
	for (int i = index; i < size; i += stride)
	{
		// ft_percentage_bar(mrt);
		x = i % (mrt->ix);
		y = i / mrt->ix;
		color = get_pixel_color(mrt, x + 1, y + 1);
		my_mlx_pixel_put(mrt, x, y, color);
		// i++;
		// printf("ix: %d / %d\r", i, mrt->ix);
	}
}
//     0  1  2  3

// 0   0  1  2  3
// 1   4  5  6  7
// 2   8  9  10 11

void ft_paint_host(t_mrt *mrt)
{
	int		i;
	int		j;
	int		size = mrt->ix * mrt->iy;
	int		color;

	// i = 0;
	// // printf("mrt: %p\n", mrt);
	// while (i < mrt->ix)
	// {
	// 	j = 0;
	// 	// ft_percentage_bar(mrt);
	// 	while (j < mrt->iy - 1)
	// 	{
	// 		color = get_pixel_color(mrt, i + 1, j + 1);
	// 		my_mlx_pixel_put(mrt, i, j, color);
	// 		j++;
	// 	}
	// 	i++;
	// 	// printf("ix: %d / %d\r", i, mrt->ix);
	// }
	int x, y = 0;
	for (int i = 0; i < size; i += 1)
	{
		// ft_percentage_bar(mrt);
		x = i % (mrt->ix);
		y = i / mrt->ix;
		color = get_pixel_color(mrt, x + 1, y + 1);
		my_mlx_pixel_put(mrt, x, y, color);
		// i++;
		// printf("ix: %d / %d\r", i, mrt->ix);
	}
	// return (NULL);
}

void	pixel_calcul(t_mrt *mrt)
{
	int deviceId;
	int numberOfSMs;

	CUDA_CALL(hipGetDevice(&deviceId));
	CUDA_CALL(hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId));
	
	size_t threadsPerBlock = 32 * 3;
	size_t numberOfBlocks = 5 * numberOfSMs;
	
	// t_mrt	*dat;
	// init_minirt(dat, "scenes/mirror_balls.rt", 1);
	// dat = ft_copy_mrt(mrt);
	// printf("\nHELLO\n");
	// CUDA_CALL(hipMemPrefetchAsync((void *)mrt->addr, sizeof(mrt->addr), deviceId));
	// for (int i = 0; i < 10; i++)
		// ft_paint_device<<<threadsPerBlock, numberOfBlocks>>>(mrt, mrt->ix * mrt->iy);
	CUDA_CALL(hipGetLastError());
	for (int i = 0; i < 10; i++)
		ft_paint_host(mrt);
	CUDA_CALL(hipDeviceSynchronize());
	// int		i;

	// i = 0;
	// mrt->i = 0;
	// while (i < THRE#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("%s\n",hipGetErrorString(x)); \
    system("pause"); \
    return EXIT_FAILURE;}} while(0)ADS)
	// {
	// 	dat[i].i = i;
	// 	pthread_create(&mrt->threads[i], NULL, \
	// 	(void *)ft_paint, (void *)&dat[i]);
	// 	i++;
	// }
	// i = -1;
	// while (++i < THREADS)
	// 	pthread_join(mrt->threads[i], NULL);
	// if (mrt->first)
	// 	printf("\r[100%%]\n");
	// ft_free_mrt(dat, THREADS);
}
