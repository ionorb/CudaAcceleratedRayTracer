#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   paint.c                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/02/21 22:24:35 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/07 18:23:52 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

void	my_mlx_pixel_put(t_mrt *mrt, int x, int y, int color)
{
	char	*dst;

	// if (!mrt->save && x < BORDER)
		// color = diminish_color(color, 0.3);
	dst = mrt->addr + (y * mrt->sizel + x * (mrt->bpp / 8));
	*(unsigned int *)dst = color;
}

void __global__ ft_paint_device(t_mrt *mrt)
{
	// t_mrt	*mrt;
	int		i;
	int		j;
	int		color;

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	(void)index;
	(void)stride;
	// mrt = (t_mrt *)data;
	i = 0;
	printf("\nHELLO: %d\n", index);
	printf("mrt: %p\n", mrt);
	// mrt->ix = 1000;
	// mrt->iy = 1800;
	while (i < mrt->ix)
	{
		j = 0;
		// ft_percentage_bar(mrt);
		while (j < mrt->iy - 1)
		{
			color = get_pixel_color(mrt, i + 1, j + 1);
			my_mlx_pixel_put(mrt, i, j, color);
			j++;
		}
		i++;
	}
	// return (NULL);
}

void ft_paint_host(void *data)
{
	t_mrt	*mrt;
	int		i;
	int		j;
	int		color;

	mrt = (t_mrt *)data;
	i = 0;
	// printf("mrt: %p\n", mrt);
	// mrt->ix = 1000;
	// mrt->iy = 1800;
	while (i < mrt->ix)
	{
		j = 0;
		// ft_percentage_bar(mrt);
		while (j < mrt->iy - 1)
		{
			color = get_pixel_color(mrt, i + 1, j + 1);
			my_mlx_pixel_put(mrt, i, j, color);
			j++;
		}
		i++;
		printf("ix: %d / %d\r", i, mrt->ix);
	}
	// return (NULL);
}

void	pixel_calcul(t_mrt *mrt)
{
	int deviceId;
	int numberOfSMs;

	CUDA_CALL(hipGetDevice(&deviceId));
	CUDA_CALL(hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId));
	
	size_t threadsPerBlock = 1;//256;
	size_t numberOfBlocks = 1;//32 * numberOfSMs;
	
	// t_mrt	*dat;
	// init_minirt(dat, "scenes/mirror_balls.rt", 1);
	// dat = ft_copy_mrt(mrt);
	// printf("\nHELLO\n");
	
	ft_paint_device<<<threadsPerBlock, numberOfBlocks>>>(mrt);
	CUDA_CALL(hipGetLastError());
	// ft_paint_host(mrt);
	CUDA_CALL(hipDeviceSynchronize());
	// int		i;

	// i = 0;
	// mrt->i = 0;
	// while (i < THRE#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("%s\n",hipGetErrorString(x)); \
    system("pause"); \
    return EXIT_FAILURE;}} while(0)ADS)
	// {
	// 	dat[i].i = i;
	// 	pthread_create(&mrt->threads[i], NULL, \
	// 	(void *)ft_paint, (void *)&dat[i]);
	// 	i++;
	// }
	// i = -1;
	// while (++i < THREADS)
	// 	pthread_join(mrt->threads[i], NULL);
	// if (mrt->first)
	// 	printf("\r[100%%]\n");
	// ft_free_mrt(dat, THREADS);
}
