#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   sphere_bump.c                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: gamoreno <gamoreno@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/03/30 02:50:57 by gamoreno          #+#    #+#             */
/*   Updated: 2023/04/08 12:59:53 by gamoreno         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

t_vec	sph_normal_from_map(t_mrt *mrt, t_inter inter, t_vec sph_coor)
{
	int		bump_coor[2];
	t_vec	ret;
	double	pol_res;
	double	as_res;

	pol_res = PI / mrt->sphere[inter.index].option.bump_map.height;
	as_res = (2 * PI) / (mrt->sphere[inter.index].option.bump_map.width - 1);
	bump_coor[0] = (int)integer_part(sph_coor.y / pol_res);
	bump_coor[1] = (int)integer_part(sph_coor.z / as_res);
	pthread_mutex_lock(mrt->mutexs);
	ret = bump_nrml_by_coor(&mrt->sphere[inter.index].option, \
	bump_coor[0], bump_coor[1], 0.05);
	pthread_mutex_unlock(mrt->mutexs);
	return (ret);
}

t_base	sph_get_tang_base(t_vec	sph_coor, t_vec z)
{
	t_base	ret;

	ret.bs_orig = fill_coord(0, 0, 0);
	ret.n1 = fill_coord(sin(sph_coor.y), cos(sph_coor.y), 0);
	ret.n3 = z;
	ret.n2 = normalize(cross_prod(ret.n3, ret.n1));
	return (ret);
}

t_vec	sphere_bumped(t_mrt *mrt, t_inter inter, t_vec without)
{
	t_mtrx	chg;
	t_vec	new_inter;
	t_vec	new_normal;
	t_vec	ret;
	t_base	tang_base;

	chg = fill_mtrx(mrt->sphere[inter.index].base.n1,
			mrt->sphere[inter.index].base.n2,
			mrt->sphere[inter.index].base.n3);
	new_inter = vec_rest(inter.inter_coor, mrt->sphere[inter.index].center);
	new_inter = mtrx_by_vec(chg, new_inter);
	new_inter = get_spheric_coord(new_inter);
	new_normal = sph_normal_from_map(mrt, inter, new_inter);
	tang_base = sph_get_tang_base(new_inter, mtrx_by_vec(chg, without));
	ret = get_bump_nrml(new_normal, tang_base, mtrx_trsp(chg));
	if (dot_prod(ret, without) < 0)
		ret = normalize(scal_vec(-1, ret));
	return (ret);
}
