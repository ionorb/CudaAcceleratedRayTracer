/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   memory_utils.c                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: yridgway <yridgway@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/02/06 01:36:18 by ionorb            #+#    #+#             */
/*   Updated: 2023/04/05 03:30:48 by yridgway         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../../includes/minirt.h"

void	ft_save_mlx(void *ptr, void **mlx, void **win, void **img)
{
	t_mrt	*mrt;

	mrt = (t_mrt *)(ptr);
	*mlx = mrt->mlx;
	*win = mrt->win;
	*img = mrt->img;
}

void	ft_free_mlx(void **mlx, void **win, void **img)
{
	if (*img)
		mlx_destroy_image(*mlx, *img);
	if (*win)
		mlx_destroy_window(*mlx, *win);
	if (*mlx)
	{
		mlx_destroy_display(*mlx);
		hipFree(*mlx);
	}
}

int	ft_free_one(t_mem *mem, void *thing)
{
	t_mem	*prev;
	t_mem	*after;

	if (!mem || !thing)
		return (0);
	prev = mem;
	if (mem)
		mem = mem->next;
	while (mem && mem->next)
	{
		after = mem->next;
		if (mem->ptr == thing || !mem->ptr)
		{
			hipFree(mem->ptr);
			hipFree(mem);
			mem = after;
			prev->next = after;
		}
		prev = mem;
		mem = mem->next;
	}
	if (mem && mem->ptr == thing)
		return (hipFree(mem->ptr), hipFree(mem), prev->next = NULL, 0);
	return (0);
}

int	mem_size(t_mem *mem)
{
	int	i;

	i = 0;
	while (mem)
	{
		i++;
		mem = mem->next;
	}
	return (i);
}
