/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   memory.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: yridgway <yridgway@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2023/01/30 15:34:04 by yridgway          #+#    #+#             */
/*   Updated: 2023/04/05 03:38:18 by yridgway         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "minirt.h"

void	*ft_liberate(t_mem *mem, int type)
{
	t_mem	*prev;

	while (mem)
	{
		if (mem->ptr)
			hipFree(mem->ptr);
		prev = mem;
		mem = mem->next;
		hipFree(prev);
	}
	if (type == EXIT_ERROR)
		exit(1);
	else if (type == EXIT_OK)
		exit(0);
	return (NULL);
}

t_mem	*mem_addback(t_mem **mem, t_mem *newy)
{
	t_mem	*tmp;

	tmp = *mem;
	if (!newy)
		return (ft_memory(NULL, EXIT_ERROR), (t_mem*)NULL);
	if (!tmp)
		return (mem = &newy, *mem);
	while (tmp && tmp->next)
		tmp = tmp->next;
	tmp->next = newy;
	return (*mem);
}

t_mem	*mem_new(size_t size, void *thing)
{
	t_mem	*newy;
	char	*err;
	hipError_t cudaErr;

	err = "Error: malloc failed\n";
	// newy = (t_mem*)malloc(sizeof(t_mem));
	cudaErr = hipMallocManaged((void **)&newy, sizeof(t_mem));
	if (!newy)
	{
		printf("cudaErr: %s\n", hipGetErrorString(cudaErr));
		return (ft_putstr_fd(err, 2), (t_mem*)NULL);
	}
	if (thing)
		newy->ptr = thing;
	else
	{
		hipMallocManaged(&newy->ptr, size);
		// newy->ptr = malloc(size);
	}
	if (!newy->ptr)
	{
		ft_putstr_fd(err, 2);
		hipFree(newy);
		newy = NULL;
		return (NULL);
	}
	newy->next = NULL;
	return (newy);
}

void	ft_close_fd(int *fd)
{
	if (*fd > 2)
	{
		close(*fd);
		*fd = -2;
	}
	ft_memory(fd, SAVE_FD);
}

void	*ft_memory(void *ptr, long long int size)
{
	static t_mem	*mem = NULL;
	t_mem			*newy;
	// static void		*mlx[3];
	static int		fd;

	if (size == MEM_SIZE)
		return (printf("mem size: %d\n", mem_size(mem)), (void*)NULL);
	if (size == FREE_ONE)
		return (ft_free_one(mem, ptr), (void*)NULL);
	if (ptr && size == ADD_TO_MEM)
		return (mem = mem_addback(&mem, mem_new(0, ptr)));
	if (size == EXIT_ERROR || size == EXIT_OK)
		return (ft_close_fd(&fd),
			mem = (t_mem*)ft_liberate(mem, size), (void*)NULL);
	// if (ptr && size == SAVE_MLX)
	// 	return (ft_save_mlx(ptr, &mlx[0], &mlx[1], &mlx[2]), (void*)NULL);
	if (ptr && size == SAVE_FD)
		return (fd = *(int *)ptr, (void*)NULL);
	newy = mem_new(size, NULL);
	mem = mem_addback(&mem, newy);
	return (newy->ptr);
}
